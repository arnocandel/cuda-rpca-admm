#include "hip/hip_runtime.h"
#include "rpca_admm_kernel.cuh"


__global__ void nodiag_normalize(float *A, float *I, int n, int i){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n)
	if (x == i && x!=y){
		I[x*n + y] /= A[i*n + i];
		A[x*n + y] /= A[i*n + i];
	}
	
}

__global__ void diag_normalize(float *A, float *I, int n, int i){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n)
	if (x == y && x == i){
		I[x*n + y] /= A[i*n + i];
		A[x*n + y] /= A[i*n + i];
	}

}

__global__ void gaussjordan(float *A, float *I, int n, int i)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n){
		if (x != i){
			I[x*n + y] -= I[i*n + y] * A[x*n + i];
			if (y != i){
				A[x*n + y] -= A[i*n + y] * A[x*n + i];
			}	 
		}
	}

}

__global__ void set_zero(float *A, float *I, int n, int i){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n){
		if (x != i){
			if (y == i){
				A[x*n + y] = 0;
			}
		}
	}
}
